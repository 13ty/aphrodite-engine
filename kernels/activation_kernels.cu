#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

namespace aphrodite {

template<typename T>
__device__ __forceinline__ T silu(const T& x) {
    // x * sigmoid(x)
    return (T) (((float) x) / (1.0f + expf((float) - x)));
}

template<typename scalar_t>
__global__ void silu_and_mul_kernel(
    scalar_t* __restrict__ out,             // [num_tokens, d]
    const scalar_t* __restrict__ input,     // [num_tokens, 2, d]
    const int d) {
    const itn token_idx = blockIdx.x;
    for (int idx = threadIdx.x; idx < d; idx += blockDim.x) {
        const scalar_t x = __ldg(&input[token_idx * 2 * d + idx]);
        const scalar_t y = __ldg(&input[token_idx * 2 * d + d + idx]);
        out[token_idx * d + idx] = silu(x) * y;
    }
}

} // namespace aphrodite

void silu_and_mul(
    torch::Tensor& out,             // [num_tokens, d]
    torch::Tensor& input)           // [num_tokens, 2 * d]
{
    int num_tokens = input.size(0);
    int d = input.size(1) / 2;

    dim3 grid(num_tokens);
    dim3 block(std::min(d, 1024));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "silu_and_mul_kernel",
        [&] {
        aphrodite::silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            d);
        });
}
