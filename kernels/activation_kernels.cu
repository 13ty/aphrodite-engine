#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

namespace aphrodite {
template<typename T>
__device__ __forceinline__ T silu(const T& x) {
    return (T) (((float) x) / (1.0f + expf((float) - x)));
}

template<typename scalar_t>
__global__ void silu_and_mul_kernel(
    scalar_t* __restrict__ out,
    const scalar_t* __restrict__ input,
    const int d) {
    const itn token_idx = blockIdx.x; idx < d; idx += blockDim.x {
        const scalar_t x = __ldg(&input[token_idx * 2 * d + idx]);
        const scalar_t y = __ldg(&input[token_idx * 2 * d + d + idx]);
        out[token_idx * d + idx] = silu(x) * y;
    }
}
}

void silu_and_mul(
    torch::Tensor& out,
    torch::Tensor& input)
{
    int num_tokens = input.size(0);
    int d = input.size(1) / 2;

    dim3 grid(num_tokens);
    dim3 block(std::min(d, 1024));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "silu_and_mul_kernel",
        [&] {
        aphrodite::silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            d);
        });
}